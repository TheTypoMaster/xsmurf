#include "hip/hip_runtime.h"
#include <reduction_kernel.cu>
////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
void reduce(int size, int threads, int blocks, int whichKernel, int *d_idata, int *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = threads * sizeof(int);

	// choose which of the optimized versions of reduction to launch
    switch (whichKernel)
    {
    case 0:
        reduce0<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
        break;
    case 1:
        reduce1<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
        break;
    case 2:
        reduce2<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
        break;
    case 3:
        reduce3<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
        break;
    case 4:
        reduce4<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
        break;
    case 5:
    default:
        switch (threads)
        {
        case 512:
            reduce5<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 256:
            reduce5<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 128:
            reduce5<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 64:
            reduce5< 64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 32:
            reduce5< 32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 16:
            reduce5< 16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  8:
            reduce5<  8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  4:
            reduce5<  4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  2:
            reduce5<  2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  1:
            reduce5<  1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        }
        break;       
    case 6:
        switch (threads)
        {
        case 512:
            reduce6<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            reduce6<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            reduce6<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            reduce6< 64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            reduce6< 32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            reduce6< 16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            reduce6<  8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            reduce6<  4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            reduce6<  2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            reduce6<  1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
        break;       
    }
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel 
// 6, we observe the maximum specified number of blocks, because each thread in 
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
    if (whichKernel < 3)
    {
        threads = (n < maxThreads) ? n : maxThreads;
        blocks = n / threads;
    }
    else
    {
        if (n == 1) 
            threads = 1;
        else
            threads = (n < maxThreads*2) ? n / 2 : maxThreads;
        blocks = n / (threads * 2);

        if (whichKernel == 6)
            blocks = min(maxBlocks, blocks);
    }
}



////////////////////////////////////////////////////////////////////////////////
// This function performs a reduction of the input data multiple times and 
// measures the average reduction time.
////////////////////////////////////////////////////////////////////////////////
int computeSum(int  n, 
	       int  numThreads,
	       int  numBlocks,
	       int  maxThreads,
	       int  maxBlocks,
	       int  whichKernel, 
	       int* d_idata)
{
    int gpu_result = 0.0f;
    int *d_odata;

    CUDA_SAFE_CALL( hipMalloc((void**) &d_odata, numBlocks*sizeof(int)) );

    gpu_result = 0.0f;
    
    // execute the kernel
    reduce(n, numThreads, numBlocks, whichKernel, d_idata, d_odata);
    
    // check if kernel execution generated an error
    CUT_CHECK_ERROR("Kernel execution failed");
    
    // sum partial block sums on GPU
    int s=numBlocks;
    int kernel = (whichKernel == 6) ? 5 : whichKernel;
    while(s > 1) 
      {
	int threads = 0, blocks = 0;
	getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);
	reduce(s, threads, blocks, kernel, d_odata, d_odata);
	if (kernel < 3)
	  s = s / threads;
	else
	  s = s / (threads*2);
      }
    CUDA_SAFE_CALL( hipMemcpy( &gpu_result, d_odata, sizeof(int), hipMemcpyDeviceToHost) );
    //gpu_result = d_odata[0];

    CUDA_SAFE_CALL(hipFree(d_odata));

    return gpu_result;
}

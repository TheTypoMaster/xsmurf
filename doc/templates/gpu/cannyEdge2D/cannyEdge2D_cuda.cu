#include "hip/hip_runtime.h"
/**************************************************
 * cannyEdge2D_cuda.c - program of 2D edges detection
 *                                        run on host and device (GPU)
 *
 *
 * DESCRIPTION:
 * programme test pour port vers GPU (NVIDIA/CUDA)
 * et benchmark
 *
 * P. Kestener (CEA, IRFU/SEDI, March 2008)
 **************************************************/

#include <stdio.h>
#include <malloc.h>
#include <fcntl.h> /* open, close */
#include <sys/stat.h> /* open, close */
#include <sys/types.h> /* open, close */
#include <unistd.h> /* write */
#include <stdlib.h> /* to make 'atof' useable */
#include <string.h>

#include <time.h>

#include <math.h>

#include <getopt.h>
#include <matheval.h>

/* parse command line arguments */
#include "cmd_cannyEdge2D_cuda.h"

/*
 * calling command line
 *
 * ./cannyEdge2D_cuda -o edge -i image.xsm
 *  ../../bin/linux/release/cannyEdge2D_cuda -i image.xsm -o edge
 *
 * or
 *
 * ./cannyEdge2D_cuda --output edge --input image.xsm
 *
 */

/* 
 * CUDA header
 */ 
// includes, project
#include <hipfft/hipfft.h>
#include <cutil.h>

// Complex data type
typedef float2 Complex; 


/* custom header */
#include "fft_utils.h"
/*#include "edge.h"
#include "misc.h"*/

#define FORWARD 0
#define BACKWARD 1


// Thread block size
//#define BLOCK_SIZE 8


/* kernel functions */
#include "cannyEdge2D_cuda_kernel.cu"
#include <reduction.cu>



int main( int argc, char* argv[] )
{
  /* data buffers */
  FILE *fileIn;

  /* buffer for gradient components */
  /*float *gradx, *grady;
    float *mod, *arg, *max;*/

  /* input image parameters */
  int thesize,lx,ly,Lx,Ly;
  unsigned int type;
  char    tempBuffer[100],saveFormat[10];

  /* output */
  char outputName[80];

  /* execution time variables */
  int t0, t1;

  /* command line argument parser */
  struct gengetopt_args_info args_info;

  /* other variables */
  int i;
  float amin=1.0;
  int octN = 1;
  int voxN = 5;
  float scale;
  char scaleId[4];

  /* cuda related variables */ 
  hipError_t res;
  int BLOCK_SIZE=8;

  /* let's call our cmdline parser */
  if (cmdline_parser (argc, argv, &args_info) != 0)
    exit(EXIT_FAILURE);
  
  /* parse output parameter */
  if (strlen(args_info.output_arg) >70) {
    fprintf(stderr, "Output filename prefix is too large.\n");
    exit(EXIT_FAILURE);
  }
  strcpy(outputName, args_info.output_arg);

  if (args_info.blocksize_given)
    BLOCK_SIZE = args_info.blocksize_arg;
  if (args_info.octave_given)
    octN = args_info.octave_arg;
  if (args_info.vox_given)
    voxN = args_info.vox_arg;
  
  /*
   * open input 2D image (float data, Xsmurf format)
   */
  if (!(fileIn = fopen( args_info.input_arg , "r"))) {
    fprintf( stderr, "Couldn't open '%s' for reading.", args_info.input_arg);
    exit(EXIT_FAILURE);
  } 
  
  /*
   * read data binary xsmurf format
   */
  //fscanf(fileIn, "Binary %d %dx%d %d(%d byte reals)\n", type, lx, ly, thesize,(int) sizeof(real));
  fgets(tempBuffer, 100, fileIn);
  sscanf(tempBuffer, "%s %d %dx%d %d",
         saveFormat, &type, &lx, &ly, &thesize);
  
  /*
   * take care that we have to swap lx and ly for fftw 
   * because fftw uses row-major whereas xsmurf uses col-major
   * 
   */
  Lx = ly;
  Ly = lx;
  
  /********************
   * CUDA
   ********************/
  CUT_DEVICE_INIT();

  // display CUDA device info
  int deviceCount;
  CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
  for (int dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    printf("  Major revision number:                         %d\n", deviceProp.major);
    printf("  Minor revision number:                         %d\n", deviceProp.minor);
    printf("  Total amount of global memory:                 %d bytes\n", deviceProp.totalGlobalMem);
    printf("  Clock rate:                                    %d kilohertz\n", deviceProp.clockRate);
  }


  /*
   * PINNED memory allocation on host
   */
  Complex *dataIn;
  res=hipHostMalloc((void **) &dataIn, Lx*Ly* sizeof(Complex));
  if (res != 0) {
    fprintf(stderr,"failed to alloc host mem for dataIn\n");
    exit(EXIT_FAILURE);
  }
  
  /* reading input data */
  float datain;
  for (i=0;i<Lx*Ly;i++) {
    fread(&datain, sizeof(float), 1, fileIn);
    dataIn[i].x = datain;
    dataIn[i].y = 0.0f;
  }
  
  /* close input */
  fclose( fileIn );
  
  t0 = clock( );

  /* allocated memory on device */
  Complex *deviceFourier, *deviceGradx, *deviceGrady;
  CUDA_SAFE_CALL(hipMalloc((void**)&deviceFourier, Lx*Ly*sizeof(Complex) ));
  CUDA_SAFE_CALL(hipMalloc((void**)&deviceGradx, Lx*Ly*sizeof(Complex) ));
  CUDA_SAFE_CALL(hipMalloc((void**)&deviceGrady, Lx*Ly*sizeof(Complex) ));

  /* copy data to device */
  CUDA_SAFE_CALL(hipMemcpy(deviceFourier, dataIn, Lx*Ly*sizeof(Complex), hipMemcpyHostToDevice));

  /* create cuFFT plan */
  hipfftHandle plan;
  CUFFT_SAFE_CALL(hipfftPlan2d(&plan, Lx, Ly, HIPFFT_C2C));
        
  /* perform cuFFT */
  CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)deviceFourier, (hipfftComplex *)deviceFourier, HIPFFT_FORWARD));

  /* modulus, argument buffers on device */
  float *deviceMod, *deviceArg;
  CUDA_SAFE_CALL(hipMalloc((void**)&deviceMod, Lx*Ly*sizeof(float) ));
  CUDA_SAFE_CALL(hipMalloc((void**)&deviceArg, Lx*Ly*sizeof(float) ));
  float *deviceMaxima;
  CUDA_SAFE_CALL(hipMalloc((void**)&deviceMaxima, lx*ly*sizeof(float) ));
  
  /* some host buffer for output */
  float *maximaMask;
  float *arg;
  hipHostMalloc((void **) &maximaMask, Lx*Ly* sizeof(float));
  hipHostMalloc((void **) &arg, Lx*Ly* sizeof(float));

  /*
   * LOOP overs scales
   */
  int num=0;
  for (int oct = 0; oct < octN; oct++) {
    for (int vox = 0; vox < voxN; vox++, num++) {
      scale = 6.0/0.86*amin*powf(2,oct+(vox/float(voxN)));
      sprintf(scaleId,"%.3d",num);
      strcpy(outputName, args_info.output_arg);
      strcat(outputName,scaleId);
      printf("%s\n",outputName);

      /* copy Fourier image into gradx and grady before performing filtering */
      CUDA_SAFE_CALL(hipMemcpy(deviceGradx, deviceFourier, Lx*Ly*sizeof(Complex), hipMemcpyDeviceToDevice));
      CUDA_SAFE_CALL(hipMemcpy(deviceGrady, deviceFourier, Lx*Ly*sizeof(Complex), hipMemcpyDeviceToDevice));
      
      /* execution parameter for Fourier filtering  */
      dim3 gridSize(Lx/BLOCK_SIZE, Ly/BLOCK_SIZE);
      dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
      fft_filter_gradx_kernel<<< gridSize, blockSize >>>(deviceGradx,Lx,Ly,scale);
      fft_filter_grady_kernel<<< gridSize, blockSize >>>(deviceGrady,Lx,Ly,scale);      
      
      /*
       * perform inplace backward Fourier transform
       */
      CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)deviceGradx, (hipfftComplex *)deviceGradx, HIPFFT_BACKWARD));
      CUFFT_SAFE_CALL(hipfftExecC2C(plan, (hipfftComplex *)deviceGrady, (hipfftComplex *)deviceGrady, HIPFFT_BACKWARD));

      /*
       * Compute gradient in polar representation (modulus, argument)
       */
      polar_gradient_kernel<<< gridSize, blockSize >>>(deviceMod,deviceArg, deviceGradx, deviceGrady,Lx,Ly);
      
      /*
       * Compute 2D edges; remove point non-maxima
       */
      remove_nonmaxima_gradient2D_kernel<<< gridSize, blockSize >>>(deviceMaxima, deviceGradx, deviceGrady,deviceMod,lx,ly); 
      
      /*
       * save bufferOut in an xsmurf format
       */      
      CUDA_SAFE_CALL(hipMemcpy(maximaMask,deviceMaxima, Lx*Ly*sizeof(float), hipMemcpyDeviceToHost));
      CUDA_SAFE_CALL(hipMemcpy(arg,deviceArg, Lx*Ly*sizeof(float), hipMemcpyDeviceToHost));
      SaveBuffer4xsmurf2D_extimage(maximaMask, arg, lx, ly, outputName, scale); 
    }
  }
  
  /* cufft plan destroy */
  hipfftDestroy(plan);
  
  CUDA_SAFE_CALL(hipFree(deviceFourier));
  CUDA_SAFE_CALL(hipFree(deviceGradx));
  CUDA_SAFE_CALL(hipFree(deviceGrady));
  CUDA_SAFE_CALL(hipFree(deviceMod));
  CUDA_SAFE_CALL(hipFree(deviceArg));
  CUDA_SAFE_CALL(hipFree(deviceMaxima));
  CUDA_SAFE_CALL(hipHostFree(maximaMask));
  CUDA_SAFE_CALL(hipHostFree(arg));
  CUDA_SAFE_CALL(hipHostFree(dataIn)); 
  
  t1 = clock( );
  fprintf( stdout, "       processing time was %d microseconds.\n", t1-t0 );
  
  
  //CUT_EXIT(argc, argv);
  
}


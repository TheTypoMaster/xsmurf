#include "hip/hip_runtime.h"


/****************************************************/
__device__ float gradx_filter(float x, float y) 
{
  return y*expf(-x*x-y*y);
}

/****************************************************/
__device__ float grady_filter(float x, float y) 
{
  return x*expf(-x*x-y*y);
}

/*************************************************************************/
/*
 * Compute the complex multiplication between an image of type FFTW_R2C 
 * and a function. The image data are organized as specified in the
 * fftw lib documentation (real multi-dimensional transform)
 */
__global__ void fft_filter_gradx_kernel (Complex *cData,
					 int Lx,
					 int Ly,
					 float   scale)
{
  float     tmp, kx, ky;  
  float     i_val;
  
  // read indexes
  int i = (int) (blockIdx.x * blockDim.x + threadIdx.x);
  int j = (int) (blockIdx.y * blockDim.y + threadIdx.y);
  
  
  int ij = j + i*Ly;
  
  if (i < Lx/2) {
    kx = scale/Lx*i;
  } else {
    kx = scale/Lx*(i-Lx);
  }
  if (j < Ly/2) {
    ky = scale/Ly*j;
  } else {
    ky = scale/Ly*(j-Ly);
  }
  
  i_val = gradx_filter(kx,ky);
  
  tmp = cData[ij].x;
  
  cData[ij].x = - cData[ij].y*i_val;
  cData[ij].y = tmp              *i_val;

}

__global__ void fft_filter_grady_kernel (Complex *cData,
					 int Lx,
					 int Ly,
					 float   scale)
{
  float     tmp, kx, ky;  
  float     i_val;
  
  // read indexes
  int i = (int) (blockIdx.x * blockDim.x + threadIdx.x);
  int j = (int) (blockIdx.y * blockDim.y + threadIdx.y);
  
  
  int ij = j + i*Ly;
  
  if (i < Lx/2) {
    kx = scale/Lx*i;
  } else {
    kx = scale/Lx*(i-Lx);
  }
  if (j < Ly/2) {
    ky = scale/Ly*j;
  } else {
    ky = scale/Ly*(j-Ly);
  }
  
  i_val = grady_filter(kx,ky);
  
  tmp = cData[ij].x;
  
  cData[ij].x = - cData[ij].y*i_val;
  cData[ij].y = tmp          *i_val;

}


__global__ void polar_gradient_kernel (float *mod, float *arg,
				       Complex *gradx,
				       Complex *grady,
				       int Lx,
				       int Ly)
{
  
  // read indexes
  int i = (int) (blockIdx.x * blockDim.x + threadIdx.x);
  int j = (int) (blockIdx.y * blockDim.y + threadIdx.y);
  int ij = j + i*Ly;
  
  // modulus
  mod[ij] = sqrtf( gradx[ij].x*gradx[ij].x +  grady[ij].x*grady[ij].x );

  // argument
  arg[ij] = atan2f(grady[ij].x, gradx[ij].x);

}

/*
 *
 */
__global__ void remove_nonmaxima_gradient2D_kernel (float *max,
						    Complex *gradx,
						    Complex *grady,
						    float *mod,
						    int lx,
						    int ly)
{
  
  // read indexes
  int i = (int) (blockIdx.x * blockDim.x + threadIdx.x);
  int j = (int) (blockIdx.y * blockDim.y + threadIdx.y);
  //int ij = j + i*Ly;
  int ij = i + j*lx;

  /*
   * dimensions
   */
  int dimx = lx;
  int dimy = ly;
  int dimxMinusOne = dimx - 1;
  int dimxPlusOne = dimx + 1;
  int dimyMinusOne = dimy - 1;
  
  float _EPSILON_NORM_ = 0.0000005f;
  /*
   * epsilon value to decide of the interpolation type.
   * If one derivative's absolute value is larger than this
   * epsilon (close to one), then we use the nearest value
   * else we perform a [bi,tri]linear interpolation.
   */
  float _EPSILON_DERIVATIVE_ = 0.9995f;
  
  /* 
   * pointers
   */
  /*float *fl_pt1;
  float *fl_pt2;
  float *fl_max;
  float *fl_nor;*/
  int fl_upper_left;
  float gx,gy,norme;

  /*
   * coordinates and vector's components
   */
  float normalized_gx;
  float normalized_gy;
  float x_point_to_be_interpolated;
  float y_point_to_be_interpolated;
  int x_upper_left_corner;
  int y_upper_left_corner;
  /*
   * coefficients
   */ 
  float dx, dy, dxdy;
  float c00, c01, c10, c11;
  /*
   * modulus
   */
  float interpolated_norme;
  
  if (i == 0 || i== lx-1 || j==0 || j==ly-1) {
   /*
   * we set the image border to zero.
   * First the borders along X direction,
   * second, the borders along the Y direction.
   */
    max[ij] = 0.0f;
    return;
  }
   
  /*
   * We investigate the middle of the image.
   */
  gx = gradx[ij].x;
  gy = grady[ij].x;
  norme = mod[ij];
  
  if (norme < _EPSILON_NORM_) {
    max[ij] = 0.0f;
      return;
  }
  
  normalized_gx = gx/norme;
  normalized_gy = gy/norme;
  
  /*
   * May we use the nearest value?
   */
  if ( (-normalized_gx > _EPSILON_DERIVATIVE_) ||
       (normalized_gx > _EPSILON_DERIVATIVE_) ||
       (-normalized_gy > _EPSILON_DERIVATIVE_) ||
       (normalized_gy > _EPSILON_DERIVATIVE_) ) {
    /*
     * First point to be interpolated.
     */
    x_upper_left_corner = (int)( (float)i + normalized_gx + 0.5 );
    y_upper_left_corner = (int)( (float)j + normalized_gy + 0.5 );
    interpolated_norme = mod[x_upper_left_corner + y_upper_left_corner * dimx];
    if ( norme <= interpolated_norme ) {
      max[ij] = 0.0f;
      return;
    }
    /*
     * Second point to be interpolated.
     */
    x_upper_left_corner = (int)( (float)i - normalized_gx + 0.5 );
    y_upper_left_corner = (int)( (float)j - normalized_gy + 0.5 );
    interpolated_norme = mod[x_upper_left_corner + y_upper_left_corner * dimx];
    if ( norme < interpolated_norme ) {
      max[ij] = 0.0f;
      return;
    }
    /*
     * We found a gradient extrema.
     */
    max[ij] = norme;
    return;
  }
  /*
   * From here we perform a bilinear interpolation
   */
  
  /*
   * First point to be interpolated.
   * It is the current point + an unitary vector
   * in the direction of the gradient.
   * It must be inside the image.
   */
  x_point_to_be_interpolated = (float)i + normalized_gx;
  y_point_to_be_interpolated = (float)j + normalized_gy;
  if ( (x_point_to_be_interpolated < 0.0f) ||
       (x_point_to_be_interpolated >= dimxMinusOne) ||
       (y_point_to_be_interpolated < 0.0f) ||
       (y_point_to_be_interpolated >= dimyMinusOne) ) {
    max[ij] = 0.0f;
    return;
  }
  /* 
   * Upper left corner,
   * coordinates of the point to be interpolated
   * with respect to this corner.
   */
  x_upper_left_corner = (int)x_point_to_be_interpolated;
  y_upper_left_corner = (int)y_point_to_be_interpolated;
  dx = x_point_to_be_interpolated - (float)x_upper_left_corner;
  dy = y_point_to_be_interpolated - (float)y_upper_left_corner;
  dxdy = dx * dy;
  /* 
   * bilinear interpolation of the gradient modulus 
   * norme[x_point_to_be_interpolated, y_point_to_be_interpolated] =
   *   norme[0,0] * ( 1 - dx) * ( 1 - dy ) +
   *   norme[1,0] * ( dx ) * ( 1 - dy ) +
   *   norme[0,1] * ( 1 - dx ) * ( dy ) +
   *   norme[1,1] * ( dx ) * ( dy )
   */
  c00 = 1.0f - dx - dy + dxdy;
  c10 = dx - dxdy;
  c01 = dy - dxdy;
  c11 = dxdy;
  fl_upper_left = x_upper_left_corner + y_upper_left_corner * dimx;
  interpolated_norme = mod[fl_upper_left] * c00 +
    mod[fl_upper_left + 1] * c10 +
    mod[fl_upper_left + dimx] * c01 +
    mod[fl_upper_left + dimxPlusOne] * c11;
  /*
   * We compare the modulus of the point with the
   * interpolated modulus. It must be larger to be
   * still considered as a potential gradient extrema.
   *
   * Here, we consider that it is strictly superior.
   * The next comparison will be superior or equal.
   * This way, the extrema is in the light part of the
   * image. 
   * By inverting both tests, we can put it in the
   * dark side of the image.
   */
  if ( norme <= interpolated_norme ) {
    max[ij] = 0.0f;
    return;
  }
  /*
   * Second point to be interpolated.
   * It is the current point - an unitary vector
   * in the direction of the gradient.
   * It must be inside the image.
   */
  x_point_to_be_interpolated = (float)i - normalized_gx;
  y_point_to_be_interpolated = (float)j - normalized_gy;
  if ( (x_point_to_be_interpolated < 0.0) ||
       (x_point_to_be_interpolated >= dimxMinusOne) ||
       (y_point_to_be_interpolated < 0.0) ||
       (y_point_to_be_interpolated >= dimyMinusOne) ) {
    max[ij] = 0.0f;
    return;
  } 
  /* 
   * Upper left corner.
   */
  x_upper_left_corner = (int)x_point_to_be_interpolated;
  y_upper_left_corner = (int)y_point_to_be_interpolated;
  /* we do not recompute the coefficients
     dx = x_point_to_be_interpolated - (double)x_upper_left_corner;
     dy = y_point_to_be_interpolated - (double)y_upper_left_corner;
     dxdy = dx * dy;
  */
  /*
   * We may use the previous coefficients.
   * norme[x_point_to_be_interpolated, y_point_to_be_interpolated] =
   *   norme[0,0] * c11 +
   *   norme[1,0] * c01 +
   *   norme[0,1] * c10 +
   *   norme[1,1] * c00
   *
   * WARNING: it works only if the cases where one derivative is close
   *          to -/+ 1 are already be independently processed, else
   *          it may lead to errors.
   */
  /* we do not recompute the coefficients
     c00 = 1.0 - dx - dy + dxdy;
     c10 = dx - dxdy;
     c01 = dy - dxdy;
     c11 = dxdy;
     fl_upper_left = norme + (x_upper_left_corner + y_upper_left_corner * dimx);
     interpolated_norme = *(fl_upper_left) * c00 +
     *(fl_upper_left + 1) * c10 +
     *(fl_upper_left + dimx) * c01 +
     *(fl_upper_left + dimxPlusOne) * c11;
     */
  fl_upper_left = x_upper_left_corner + y_upper_left_corner * dimx;
  interpolated_norme = mod[fl_upper_left] * c11 +
    mod[fl_upper_left + 1] * c01 +
    mod[fl_upper_left + dimx] * c10 +
    mod[fl_upper_left + dimxPlusOne] * c00;
  /*
   * Last test to decide whether or not we 
   * have an extrema
   */
  if ( norme < interpolated_norme ) {
    max[ij] = 0.0f;
    return;
  }
  /*
   * We found a gradient extrema.
   */
  max[ij] = norme;

}

/********************************************
 * new version to compute number of maxima
 ********************************************/
__global__ void remove_nonmaxima_gradient2D_kernel2 (int *max,
						     Complex *gradx,
						     Complex *grady,
						     float *mod,
						     int lx,
						     int ly)
{
  // read indexes
  int i = (int) (blockIdx.x * blockDim.x + threadIdx.x);
  int j = (int) (blockIdx.y * blockDim.y + threadIdx.y);
  //int ij = j + i*Ly;
  int ij = i + j*lx;

  /*
   * dimensions
   */
  int dimx = lx;
  int dimy = ly;
  int dimxMinusOne = dimx - 1;
  int dimxPlusOne = dimx + 1;
  int dimyMinusOne = dimy - 1;
  
  float _EPSILON_NORM_ = 0.0000005f;
  /*
   * epsilon value to decide of the interpolation type.
   * If one derivative's absolute value is larger than this
   * epsilon (close to one), then we use the nearest value
   * else we perform a [bi,tri]linear interpolation.
   */
  float _EPSILON_DERIVATIVE_ = 0.9995f;
  
  /* 
   * pointers
   */
  /*float *fl_pt1;
  float *fl_pt2;
  float *fl_max;
  float *fl_nor;*/
  int fl_upper_left;
  float gx,gy,norme;

  /*
   * coordinates and vector's components
   */
  float normalized_gx;
  float normalized_gy;
  float x_point_to_be_interpolated;
  float y_point_to_be_interpolated;
  int x_upper_left_corner;
  int y_upper_left_corner;
  /*
   * coefficients
   */ 
  float dx, dy, dxdy;
  float c00, c01, c10, c11;
  /*
   * modulus
   */
  float interpolated_norme;
  
  if (i == 0 || i== lx-1 || j==0 || j==ly-1) {
   /*
   * we set the image border to zero.
   * First the borders along X direction,
   * second, the borders along the Y direction.
   */
    max[ij] = 0.0f;
    return;
  }
   
  /*
   * We investigate the middle of the image.
   */
  gx = gradx[ij].x;
  gy = grady[ij].x;
  norme = mod[ij];
  
  if (norme < _EPSILON_NORM_) {
    max[ij] = 0.0f;
      return;
  }
  
  normalized_gx = gx/norme;
  normalized_gy = gy/norme;
  
  /*
   * May we use the nearest value?
   */
  if ( (-normalized_gx > _EPSILON_DERIVATIVE_) ||
       (normalized_gx > _EPSILON_DERIVATIVE_) ||
       (-normalized_gy > _EPSILON_DERIVATIVE_) ||
       (normalized_gy > _EPSILON_DERIVATIVE_) ) {
    /*
     * First point to be interpolated.
     */
    x_upper_left_corner = (int)( (float)i + normalized_gx + 0.5 );
    y_upper_left_corner = (int)( (float)j + normalized_gy + 0.5 );
    interpolated_norme = mod[x_upper_left_corner + y_upper_left_corner * dimx];
    if ( norme <= interpolated_norme ) {
      max[ij] = 0.0f;
      return;
    }
    /*
     * Second point to be interpolated.
     */
    x_upper_left_corner = (int)( (float)i - normalized_gx + 0.5 );
    y_upper_left_corner = (int)( (float)j - normalized_gy + 0.5 );
    interpolated_norme = mod[x_upper_left_corner + y_upper_left_corner * dimx];
    if ( norme < interpolated_norme ) {
      max[ij] = 0.0f;
      return;
    }
    /*
     * We found a gradient extrema.
     */
    max[ij] = 1;
    return;
  }
  /*
   * From here we perform a bilinear interpolation
   */
  
  /*
   * First point to be interpolated.
   * It is the current point + an unitary vector
   * in the direction of the gradient.
   * It must be inside the image.
   */
  x_point_to_be_interpolated = (float)i + normalized_gx;
  y_point_to_be_interpolated = (float)j + normalized_gy;
  if ( (x_point_to_be_interpolated < 0.0f) ||
       (x_point_to_be_interpolated >= dimxMinusOne) ||
       (y_point_to_be_interpolated < 0.0f) ||
       (y_point_to_be_interpolated >= dimyMinusOne) ) {
    max[ij] = 0.0f;
    return;
  }
  /* 
   * Upper left corner,
   * coordinates of the point to be interpolated
   * with respect to this corner.
   */
  x_upper_left_corner = (int)x_point_to_be_interpolated;
  y_upper_left_corner = (int)y_point_to_be_interpolated;
  dx = x_point_to_be_interpolated - (float)x_upper_left_corner;
  dy = y_point_to_be_interpolated - (float)y_upper_left_corner;
  dxdy = dx * dy;
  /* 
   * bilinear interpolation of the gradient modulus 
   * norme[x_point_to_be_interpolated, y_point_to_be_interpolated] =
   *   norme[0,0] * ( 1 - dx) * ( 1 - dy ) +
   *   norme[1,0] * ( dx ) * ( 1 - dy ) +
   *   norme[0,1] * ( 1 - dx ) * ( dy ) +
   *   norme[1,1] * ( dx ) * ( dy )
   */
  c00 = 1.0f - dx - dy + dxdy;
  c10 = dx - dxdy;
  c01 = dy - dxdy;
  c11 = dxdy;
  fl_upper_left = x_upper_left_corner + y_upper_left_corner * dimx;
  interpolated_norme = mod[fl_upper_left] * c00 +
    mod[fl_upper_left + 1] * c10 +
    mod[fl_upper_left + dimx] * c01 +
    mod[fl_upper_left + dimxPlusOne] * c11;
  /*
   * We compare the modulus of the point with the
   * interpolated modulus. It must be larger to be
   * still considered as a potential gradient extrema.
   *
   * Here, we consider that it is strictly superior.
   * The next comparison will be superior or equal.
   * This way, the extrema is in the light part of the
   * image. 
   * By inverting both tests, we can put it in the
   * dark side of the image.
   */
  if ( norme <= interpolated_norme ) {
    max[ij] = 0.0f;
    return;
  }
  /*
   * Second point to be interpolated.
   * It is the current point - an unitary vector
   * in the direction of the gradient.
   * It must be inside the image.
   */
  x_point_to_be_interpolated = (float)i - normalized_gx;
  y_point_to_be_interpolated = (float)j - normalized_gy;
  if ( (x_point_to_be_interpolated < 0.0) ||
       (x_point_to_be_interpolated >= dimxMinusOne) ||
       (y_point_to_be_interpolated < 0.0) ||
       (y_point_to_be_interpolated >= dimyMinusOne) ) {
    max[ij] = 0.0f;
    return;
  } 
  /* 
   * Upper left corner.
   */
  x_upper_left_corner = (int)x_point_to_be_interpolated;
  y_upper_left_corner = (int)y_point_to_be_interpolated;
  /* we do not recompute the coefficients
     dx = x_point_to_be_interpolated - (double)x_upper_left_corner;
     dy = y_point_to_be_interpolated - (double)y_upper_left_corner;
     dxdy = dx * dy;
  */
  /*
   * We may use the previous coefficients.
   * norme[x_point_to_be_interpolated, y_point_to_be_interpolated] =
   *   norme[0,0] * c11 +
   *   norme[1,0] * c01 +
   *   norme[0,1] * c10 +
   *   norme[1,1] * c00
   *
   * WARNING: it works only if the cases where one derivative is close
   *          to -/+ 1 are already be independently processed, else
   *          it may lead to errors.
   */
  /* we do not recompute the coefficients
     c00 = 1.0 - dx - dy + dxdy;
     c10 = dx - dxdy;
     c01 = dy - dxdy;
     c11 = dxdy;
     fl_upper_left = norme + (x_upper_left_corner + y_upper_left_corner * dimx);
     interpolated_norme = *(fl_upper_left) * c00 +
     *(fl_upper_left + 1) * c10 +
     *(fl_upper_left + dimx) * c01 +
     *(fl_upper_left + dimxPlusOne) * c11;
     */
  fl_upper_left = x_upper_left_corner + y_upper_left_corner * dimx;
  interpolated_norme = mod[fl_upper_left] * c11 +
    mod[fl_upper_left + 1] * c01 +
    mod[fl_upper_left + dimx] * c10 +
    mod[fl_upper_left + dimxPlusOne] * c00;
  /*
   * Last test to decide whether or not we 
   * have an extrema
   */
  if ( norme < interpolated_norme ) {
    max[ij] = 0.0f;
    return;
  }
  /*
   * We found a gradient extrema.
   */
  max[ij] = 1;

}

